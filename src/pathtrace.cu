#include "hip/hip_runtime.h"
﻿#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include <glm/gtc/matrix_inverse.hpp>

#include "utilities.h"
#include "intersections.h"
#include "interactions.h"
#include "stream_compaction/common.h"
#include "stream_compaction/efficient.h"

 
#define USECOMPATION 1
#define USETHRUSTCOMPT 0
#define SORTBYKEY 0 && !USECOMPATION
#define CACHEFIRSTBOUNCE 1
#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
 

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static int * dev_remain_bounces = NULL;
static int * dev_indices4compact = NULL;
static int * dev_bools4compact = NULL;
static PathSegment * dev_paths_buff = NULL;
static int * dev_materialID_buff = NULL;
static int * dev_materialID_buff2 = NULL;
static ShadeableIntersection * dev_intersections_firstbounce = NULL;
void pathtraceInit(Scene *scene) {
	hst_scene = scene;
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need
	hipMalloc(&dev_remain_bounces, pixelcount * sizeof(int));
	hipMalloc(&dev_indices4compact, pixelcount * sizeof(int));
	hipMalloc(&dev_bools4compact, pixelcount * sizeof(int));
	hipMalloc(&dev_paths_buff, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_materialID_buff, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_materialID_buff2, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_intersections_firstbounce, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections_firstbounce, 0, pixelcount * sizeof(ShadeableIntersection));

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created
	hipFree(dev_remain_bounces);
	hipFree(dev_indices4compact);
	hipFree(dev_bools4compact);
	hipFree(dev_paths_buff);
	hipFree(dev_materialID_buff);
	hipFree(dev_materialID_buff2);
	hipFree(dev_intersections_firstbounce);
	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
				//when terminate, when use these values, TODO!
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
			intersections[path_index].pt3 = intersect_point; //leave it here for now
		}
	}
}

__device__ void progressGatherPath(glm::vec3 * image, const PathSegment& path_segment)
{
	if (path_segment.isoff() && USECOMPATION)
	{ //refer to final gather
		image[path_segment.pixelIndex] += path_segment.color;
	}
}

__global__ void shadeMaterialAndGather(int iter
	, int num_paths
	, int depth
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	, glm::vec3* image){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths && pathSegments[idx].remainingBounces > 0)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];

		if (intersection.t <= 0.0f){
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
		else {
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);
			//thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;
			if (material.emittance > 0.0f){
				pathSegments[idx].color *= (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			else {
				scatterRay(pathSegments[idx], intersection.pt3, intersection.surfaceNormal, material, rng);
				pathSegments[idx].remainingBounces--;
			}
		}
		progressGatherPath(image, pathSegments[idx]);
	}

}

// Add the current iteration's output to the overall image
__global__ void finalGatherDone(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		if (iterationPath.isoff() && USECOMPATION){
			image[iterationPath.pixelIndex] += iterationPath.color;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

//add path termination bool return
struct isPathOff{
	__host__ __device__ bool operator()(const PathSegment & path_seg){
		return path_seg.isoff();
	}
};

//get the remaining bounces for streamcompaction
__global__ void getPathBounces(int n, int *obounces, const PathSegment * paths){
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < n)
	{
		PathSegment pathSegment = paths[path_index];
		obounces[path_index] = pathSegment.remainingBounces;
	}
}

__global__ void kernScatterPaths(int n, PathSegment *odata,
	const PathSegment *idata, const int *bools, const int *indices) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= n) return;
	if (bools[index]){
		odata[indices[index]] = idata[index];
	}
}
__global__ void kernGetMaterialID(int n, int *obuff, const ShadeableIntersection * intersects){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index > n){
		obuff[index] = intersects[index].materialId;
	}
}
/**
* Wrapper for the __global__ call that sets up the kernel calls and does a ton
* of memory management
*/
void pathtrace(uchar4 *pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	//printf("traceDepth %d \n", traceDepth);
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	//add thrust ptr
	thrust::device_ptr<PathSegment> thrust_dev_path_ptr(dev_paths);
	//


	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.



	// TODO: perform one iteration of path tracing
	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> >(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray failed");
	//TRY motion here     ///////////////////////////
	Geom *geoms = &(hst_scene->geoms)[0];
	glm::vec3 curTrans;
	for (int i = 0; i < hst_scene->geoms.size(); i++){
		if (geoms[i].isMoving){
			curTrans = geoms[i].translation;
			curTrans = geoms[i].translation + (geoms[i].movegoal - curTrans) *  (float)0.01;
			//printf("%f \n", (geoms[i].movegoal - curTrans).x);
			//printf("%f \n", (geoms[i].movegoal - curTrans).x *  (float)0.1 );
			//printf("%f \n", curTrans.x);
			//printf("%f \n", curTrans.x);
			//printf("%f \n", geoms[i].movegoal.x);
			geoms[i].translation = curTrans;
			geoms[i].transform = utilityCore::buildTransformationMatrix(curTrans, geoms[i].rotation, geoms[i].scale);
			geoms[i].inverseTransform = glm::inverse(geoms[i].transform);
			geoms[i].invTranspose = glm::inverseTranspose(geoms[i].transform);
		}
	}
	hipMemcpy(dev_geoms, geoms, hst_scene->geoms.size()*sizeof(Geom), hipMemcpyHostToDevice);
	//end motion       /////////////////////////////

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;
	//active paths
	int num_paths_on = num_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths_on + blockSize1d - 1) / blockSize1d;
		if (CACHEFIRSTBOUNCE && depth == 0 && iter == 1 || !CACHEFIRSTBOUNCE){
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();

		}
		if (CACHEFIRSTBOUNCE){
			if (depth == 0 && iter == 1){
				hipMemcpy(dev_intersections_firstbounce, dev_intersections, num_paths_on * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}
			else if (depth == 0&&iter > 1){//we only cache the first bounce at the very beginning and reuse it as the first bounce in other iterations (assume stationary camera and scene)
				hipMemcpy(dev_intersections, dev_intersections_firstbounce, num_paths_on * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}
			else{
				computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
					depth
					, num_paths
					, dev_paths
					, dev_geoms
					, hst_scene->geoms.size()
					, dev_intersections
					);
				checkCUDAError("trace one bounce");
				hipDeviceSynchronize();
			}
		}


		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.
		int numoffbySort = 0;
#if SORTBYKEY
		kernGetMaterialID << <numblocksPathSegmentTracing, blockSize1d >> >(num_paths_on, dev_materialID_buff, dev_intersections);
		hipMemcpy(dev_materialID_buff2, dev_materialID_buff, num_paths_on*sizeof(int), hipMemcpyDeviceToDevice);
		thrust::sort_by_key(thrust::device, dev_materialID_buff, dev_materialID_buff + num_paths_on, dev_paths);
		thrust::sort_by_key(thrust::device, dev_materialID_buff2, dev_materialID_buff2 + num_paths_on, dev_intersections);
/*		numoffbySort = thrust::count_if(thrust_dev_path_ptr, thrust_dev_path_ptr + num_paths_on, isPathOff());
		num_paths_on -= numoffbySort;
		printf("num_paths_on %d\n", num_paths_on)*/;
#endif
		shadeMaterialAndGather << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth,
			num_paths_on,
			iter,
			dev_intersections,
			dev_paths,
			dev_materials,
			dev_image
			);



#if USECOMPATION
#if USETHRUSTCOMPT
		//steam compaction by thrust
		auto thrustend = thrust::remove_if(thrust::device, thrust_dev_path_ptr, thrust_dev_path_ptr + num_paths_on, isPathOff());
		num_paths_on = thrustend - thrust_dev_path_ptr;
#else
		num_paths_on = StreamCompaction::Efficient::compactPaths(num_paths_on, dev_paths_buff, dev_indices4compact, dev_bools4compact, dev_paths);
#endif
#endif
		//printf("%d \n", num_paths_on);
		
		depth++;
		iterationComplete = depth > traceDepth || num_paths_on <= 0; // DONE: should be based off stream compaction results.
		


	}

	// Assemble this iteration and apply it to the image
#if !USECOMPATION
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<< <numBlocksPixels, blockSize1d >> >(num_paths, dev_image, dev_paths);
#endif
	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> >(pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
