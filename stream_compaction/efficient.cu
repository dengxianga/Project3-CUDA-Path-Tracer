#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define NUM_BANKS 16  
#define LOG_NUM_BANKS 4  
#define CONFLICT_FREE_OFFSET(n) \
	((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

 

namespace StreamCompaction {
namespace Efficient {

// TODO: __global__
__global__ void upSweep(int offset, int n,   int *idata){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index >=n) return;
	int tmp=(offset << 1);
	if (index % tmp==0){
		if (index + tmp <=n){ 
			idata[index+tmp-1] += idata[index+offset-1]  ;		 
		}
	}
}

__global__ void downSweep(int offset, int n,  int *idata){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index >=n) return;
	int tmp=(offset << 1);
	if (index % tmp==0){

		if (index + tmp <= n){
			int t = idata[index + offset -1];
			idata[index+offset-1] = idata[index+ tmp -1];
			idata[index+ tmp -1] += t ;
		}
 
	}
}


/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
float scan(int n, int *odata, const int *idata) {
	hipEvent_t t0, t2;
	hipEventCreate(&t0);
	hipEventCreate(&t2); 
 
	float milliscs = 0.0f;
	float tmpt;
    // TODO
    //printf("TODO\n");
	int levels_max = ilog2ceil(n);
	int n_max= 1 << levels_max;

	dim3 numblocks(std::ceil((double) n_max / blockSize));
	int* idata_buff;
	//allocate more space than needed
	hipMalloc((void**)&idata_buff, n_max*sizeof(int)); 	
		checkCUDAError("hipMalloc-idata_buff-  failed!");	
	//reset all to zeros
    hipMemset(idata_buff, 0, n_max*sizeof(int));
		checkCUDAError("hipMemset-idata_buff-  failed!");	

	/// CPU -->GPU
	hipMemcpy(idata_buff,idata,n*sizeof(int),hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy-idata_buff-failed");

	hipEventRecord(t0);

	//upsweep
	for (int level=0; level <= levels_max-1; level++){
		upSweep<<<numblocks,blockSize>>>(1<<level, n_max, idata_buff);
	}

	hipEventRecord(t2);	
	hipEventSynchronize(t2);
	hipEventElapsedTime(&tmpt, t0, t2);	 
	milliscs += tmpt;


	//downsweep
	//set root x[n-1]=0
	//idata_buff[n_max-1]=0;
	hipMemset(idata_buff+n_max-1, 0,  sizeof(int));
		
	hipEventRecord(t0);

	for (int level=levels_max-1; level >=0 ; level--){
		downSweep<<<numblocks,blockSize>>>(1<<level, n_max, idata_buff);
	}

	hipEventRecord(t2);
	hipEventSynchronize(t2);	
	hipEventElapsedTime(&tmpt, t0, t2);
	milliscs += tmpt;

	/// GPU --> CPU
	hipMemcpy(odata, idata_buff, n*sizeof(int),hipMemcpyDeviceToHost);
		checkCUDAError("hipMemcpy-odata-failed");
	hipFree(idata_buff);
	return milliscs;
}


float scanOnDevice(int n, int *odata, const int *idata) {
 
	 
	int levels_max = ilog2ceil(n);
	int n_max = 1 << levels_max;

	dim3 numblocks(std::ceil((double)n_max / blockSize));
	int* idata_buff;
	//allocate more space than needed
	hipMalloc((void**)&idata_buff, n_max*sizeof(int));
	checkCUDAError("hipMalloc-idata_buff-  failed!");
	//reset all to zeros
	hipMemset(idata_buff, 0, n_max*sizeof(int));
	checkCUDAError("hipMemset-idata_buff-  failed!");

	/// GPU -->GPU
	hipMemcpy(idata_buff, idata, n*sizeof(int), hipMemcpyDeviceToDevice);
	checkCUDAError("hipMemcpy-idata_buff-failed");

 

	//upsweep
	for (int level = 0; level <= levels_max - 1; level++){
		upSweep << <numblocks, blockSize >> >(1 << level, n_max, idata_buff);
	}
	  


	//downsweep
	//set root x[n-1]=0
	//idata_buff[n_max-1]=0;
	hipMemset(idata_buff + n_max - 1, 0, sizeof(int));
	 

	for (int level = levels_max - 1; level >= 0; level--){
		downSweep << <numblocks, blockSize >> >(1 << level, n_max, idata_buff);
	}

 
	/// GPU --> GPU
	hipMemcpy(odata, idata_buff, n*sizeof(int), hipMemcpyDeviceToDevice);
	checkCUDAError("hipMemcpy-odata-failed");
	hipFree(idata_buff);
	return 0;
}
/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata, float &milliscs) {
	hipEvent_t t0, t2;
	hipEventCreate(&t0);
	hipEventCreate(&t2);

	milliscs = 0.0f;
	float tmpt;


    int n_remaing=0;
	int * idata_buff;
	int * odata_buff;
	int * bool_buff;
	int * indices_buff;

	dim3 numblocks(std::ceil((double) n/blockSize));
	//
	hipMalloc((void**)&idata_buff,n * sizeof(int));
		checkCUDAError("hipMalloc-idata_buff-failed");
	hipMalloc((void**)&odata_buff,n * sizeof(int));
		checkCUDAError("hipMalloc-odata_buff-failed");
	hipMalloc((void**)&bool_buff,n * sizeof(int));
		checkCUDAError("hipMalloc-odata_buff-failed");
	hipMalloc((void**)&indices_buff,n * sizeof(int));
		checkCUDAError("hipMalloc-odata_buff-failed");

	hipMemcpy(idata_buff, idata, n* sizeof(int), hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy-idata_buff-failed");
	hipMemcpy(odata_buff, odata, n* sizeof(int), hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy-odata_buff-failed");
	
	hipEventRecord(t0);
	//produce the indices
	StreamCompaction::Common::kernMapToBoolean<<<numblocks, blockSize>>> ( n, bool_buff, idata_buff);

	scan  (n, indices_buff, bool_buff);

	StreamCompaction::Common::kernScatter<<<numblocks, blockSize>>>( n, odata_buff, idata_buff,  bool_buff,  indices_buff);
	
	hipEventRecord(t2);
	hipEventSynchronize(t2);	
	hipEventElapsedTime(&tmpt, t0, t2);
	milliscs += tmpt;

	//GPU-->CPU
	hipMemcpy(odata,odata_buff,n*sizeof(int),hipMemcpyDeviceToHost);

	//for (int i =0; i< n; i++){
	//	n_remaing+=bool_buff[i];
	//}
	hipMemcpy(&n_remaing,indices_buff+n-1,sizeof(int),hipMemcpyDeviceToHost);
	int extra;
	hipMemcpy(&extra, bool_buff + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(idata_buff);
	hipFree(odata_buff);
	hipFree(bool_buff);
	hipFree(indices_buff);
	return n_remaing + extra;
}
 

__global__ void kernMapPathsToBoolean(int n, int *bools, const PathSegment *paths) {
	// TODO
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n){
		bools[index] = paths[index].remainingBounces != 0;
	}
}
__global__ void kernPathsScatter(int n, PathSegment *odata,
	const PathSegment *idata, const int *bools, const int *indices) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= n) return;
	if (bools[index]){
		odata[indices[index]] = idata[index];
	}
}
 



 

// see http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html
__global__ void preScanShared(int n, int * g_odata, const int* g_idata){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	 
	extern __shared__ float s_idata[];  // allocated on invocation  
	int thid = threadIdx.x;
	int offset = 1;
	//int ai0 = thid;
	//int bi0 = thid + (n / 2);
	//int bankOffsetA = CONFLICT_FREE_OFFSET(ai0);
	//int bankOffsetB = CONFLICT_FREE_OFFSET(bi0);
 
	s_idata[2 * thid] = g_idata[2 * index]; // load input into shared memory  
	s_idata[2 * thid + 1] = g_idata[2 * index + 1];
	//s_idata[ai0 + bankOffsetA] = g_idata[2 * index]; // load input into shared memory  
	//s_idata[bi0 + bankOffsetB] = g_idata[2 * index + 1];
	for (int d = n >> 1; d > 0; d >>= 1)                    // build sum in place up the tree  
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;
			////banking conflict
			//ai += CONFLICT_FREE_OFFSET(ai);
			//bi += CONFLICT_FREE_OFFSET(bi);

			s_idata[bi] += s_idata[ai];
		}
		offset *= 2;
		
	}
	//if (thid == 0) { s_idata[n - 1] = 0; } // clear the last element  
	if (thid == 0) { s_idata[n - 1+CONFLICT_FREE_OFFSET(n - 1)] = 0; } // clear the last element  
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan  
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;
			//banking conflict
			//ai += CONFLICT_FREE_OFFSET(ai);
			//bi += CONFLICT_FREE_OFFSET(bi);
			float t = s_idata[ai];
			s_idata[ai] = s_idata[bi];
			s_idata[bi] += t;
		}
	}
	__syncthreads();
	g_odata[2 * index] = s_idata[2 * thid]; // write results to device memory  
	g_odata[2 * index + 1] = s_idata[2 * thid + 1];
	//banking conflict	 
	//g_odata[2 * index] = s_idata[ai0 + bankOffsetA]; // write results to device memory  
	//g_odata[2 * index + 1] = s_idata[bi0 + bankOffsetB];
}
 
//http://www.eecs.umich.edu/courses/eecs570/hw/parprefix.pdf
__global__ void storeBlockSums(int n, int *sum_buff, const int * odata, const int * idata){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n){
		int offset = (index + 1)*blockSize - 1;
		sum_buff[index] = odata[offset] + idata[offset];
	}
}
__global__ void sumBuff2Blocks(int n, int *odata, int * sumbuff){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n){
		odata[index] += sumbuff[blockIdx.x];
	}
}
void scanMultiBlocks(int n, int * odata, const int * idata){
	int numblocks(std::ceil((double)n / blockSize)); 
	//int numblockshalf = numblocks / 2;
	int nmax = numblocks*blockSize;
 
	//kernel scan here
	//if (numblocks == 1){
	preScanShared << <numblocks, blockSize / 2, blockSize*sizeof(int) >> >(blockSize, odata, idata);
		//return;
	//}
	
	if (numblocks > 1){
		int numblocksum(std::ceil((double)numblocks / blockSize)); 
		int * dev_sum_buff;
		//int * dev_odata_buff;
		int *dev_scan_sum_buff;
		hipMalloc((void**)&dev_sum_buff, numblocks*sizeof(int));
		hipMalloc((void**)&dev_scan_sum_buff, numblocks *sizeof(int));

		
		storeBlockSums << <numblocksum, blockSize >> >(numblocks, dev_sum_buff, odata, idata);
		//resursive scan here
		//scanMultiBlocks(numblocksum, dev_scan_sum_buff, dev_sum_buff);
		scanMultiBlocks(numblocks, dev_scan_sum_buff, dev_sum_buff);
		sumBuff2Blocks <<< numblocks, blockSize >> > (nmax, odata, dev_scan_sum_buff);

		//free buff here
		hipFree(dev_sum_buff);
		hipFree(dev_scan_sum_buff);
	}

	
}
  

int compactPaths(int n, PathSegment * odata_buff, int * bool_buff, int * indices_buff, PathSegment *paths){
 


	int n_remaing = 0;
 
	dim3 numblocks(std::ceil((double)n / blockSize));
	//
 
 

	 
	hipMemcpy(odata_buff, paths, n* sizeof(PathSegment), hipMemcpyDeviceToDevice);
	checkCUDAError("hipMemcpy-odata_buff-failed");

	 ;
	//produce the indices
	kernMapPathsToBoolean << <numblocks, blockSize >> > (n, bool_buff, paths);

	 //scanOnDevice(n, indices_buff, bool_buff); 
	scanMultiBlocks(n, indices_buff, bool_buff);
	//int levels_max = ilog2ceil(n);
	//int n_max = 1 << levels_max;
	//dim3 numblocksmax(std::ceil((double)n_max / blockSize));
	//kernScanShared << <numblocksmax, blockSize >> >(n, indices_buff, bool_buff);

	kernPathsScatter << <numblocks, blockSize >> >(n, odata_buff, paths, bool_buff, indices_buff);

 

	//GPU-->GPU
	hipMemcpy(paths, odata_buff, n*sizeof(PathSegment), hipMemcpyDeviceToDevice);

 
	hipMemcpy(&n_remaing, indices_buff + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	int extra;
	hipMemcpy(&extra, bool_buff + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	 
 
	return n_remaing + extra;
}
}
}
