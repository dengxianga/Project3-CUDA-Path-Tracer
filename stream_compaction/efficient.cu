#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

// TODO: __global__
__global__ void upSweep(int offset, int n,   int *idata){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index >=n) return;
	int tmp=(offset << 1);
	if (index % tmp==0){
		if (index + tmp <=n){ 
			idata[index+tmp-1] += idata[index+offset-1]  ;		 
		}
	}
}

__global__ void downSweep(int offset, int n,  int *idata){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if (index >=n) return;
	int tmp=(offset << 1);
	if (index % tmp==0){

		if (index + tmp <= n){
			int t = idata[index + offset -1];
			idata[index+offset-1] = idata[index+ tmp -1];
			idata[index+ tmp -1] += t ;
		}
 
	}
}


/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
float scan(int n, int *odata, const int *idata) {
	hipEvent_t t0, t2;
	hipEventCreate(&t0);
	hipEventCreate(&t2); 
 
	float milliscs = 0.0f;
	float tmpt;
    // TODO
    //printf("TODO\n");
	int levels_max = ilog2ceil(n);
	int n_max= 1 << levels_max;

	dim3 numblocks(std::ceil((double) n_max / blockSize));
	int* idata_buff;
	//allocate more space than needed
	hipMalloc((void**)&idata_buff, n_max*sizeof(int)); 	
		checkCUDAError("hipMalloc-idata_buff-  failed!");	
	//reset all to zeros
    hipMemset(idata_buff, 0, n_max*sizeof(int));
		checkCUDAError("hipMemset-idata_buff-  failed!");	

	/// CPU -->GPU
	hipMemcpy(idata_buff,idata,n*sizeof(int),hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy-idata_buff-failed");

	hipEventRecord(t0);

	//upsweep
	for (int level=0; level <= levels_max-1; level++){
		upSweep<<<numblocks,blockSize>>>(1<<level, n_max, idata_buff);
	}

	hipEventRecord(t2);	
	hipEventSynchronize(t2);
	hipEventElapsedTime(&tmpt, t0, t2);	 
	milliscs += tmpt;


	//downsweep
	//set root x[n-1]=0
	//idata_buff[n_max-1]=0;
	hipMemset(idata_buff+n_max-1, 0,  sizeof(int));
		
	hipEventRecord(t0);

	for (int level=levels_max-1; level >=0 ; level--){
		downSweep<<<numblocks,blockSize>>>(1<<level, n_max, idata_buff);
	}

	hipEventRecord(t2);
	hipEventSynchronize(t2);	
	hipEventElapsedTime(&tmpt, t0, t2);
	milliscs += tmpt;

	/// GPU --> CPU
	hipMemcpy(odata, idata_buff, n*sizeof(int),hipMemcpyDeviceToHost);
		checkCUDAError("hipMemcpy-odata-failed");
	hipFree(idata_buff);
	return milliscs;
}
float scanOnDevice(int n, int *odata, const int *idata) {
 
	 
	int levels_max = ilog2ceil(n);
	int n_max = 1 << levels_max;

	dim3 numblocks(std::ceil((double)n_max / blockSize));
	int* idata_buff;
	//allocate more space than needed
	hipMalloc((void**)&idata_buff, n_max*sizeof(int));
	checkCUDAError("hipMalloc-idata_buff-  failed!");
	//reset all to zeros
	hipMemset(idata_buff, 0, n_max*sizeof(int));
	checkCUDAError("hipMemset-idata_buff-  failed!");

	/// CPU -->GPU
	hipMemcpy(idata_buff, idata, n*sizeof(int), hipMemcpyDeviceToDevice);
	checkCUDAError("hipMemcpy-idata_buff-failed");

 

	//upsweep
	for (int level = 0; level <= levels_max - 1; level++){
		upSweep << <numblocks, blockSize >> >(1 << level, n_max, idata_buff);
	}
	  


	//downsweep
	//set root x[n-1]=0
	//idata_buff[n_max-1]=0;
	hipMemset(idata_buff + n_max - 1, 0, sizeof(int));
	 

	for (int level = levels_max - 1; level >= 0; level--){
		downSweep << <numblocks, blockSize >> >(1 << level, n_max, idata_buff);
	}

 
	/// GPU --> GPU
	hipMemcpy(odata, idata_buff, n*sizeof(int), hipMemcpyDeviceToDevice);
	checkCUDAError("hipMemcpy-odata-failed");
	hipFree(idata_buff);
	return 0;
}
/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata, float &milliscs) {
	hipEvent_t t0, t2;
	hipEventCreate(&t0);
	hipEventCreate(&t2);

	milliscs = 0.0f;
	float tmpt;


    int n_remaing=0;
	int * idata_buff;
	int * odata_buff;
	int * bool_buff;
	int * indices_buff;

	dim3 numblocks(std::ceil((double) n/blockSize));
	//
	hipMalloc((void**)&idata_buff,n * sizeof(int));
		checkCUDAError("hipMalloc-idata_buff-failed");
	hipMalloc((void**)&odata_buff,n * sizeof(int));
		checkCUDAError("hipMalloc-odata_buff-failed");
	hipMalloc((void**)&bool_buff,n * sizeof(int));
		checkCUDAError("hipMalloc-odata_buff-failed");
	hipMalloc((void**)&indices_buff,n * sizeof(int));
		checkCUDAError("hipMalloc-odata_buff-failed");

	hipMemcpy(idata_buff, idata, n* sizeof(int), hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy-idata_buff-failed");
	hipMemcpy(odata_buff, odata, n* sizeof(int), hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy-odata_buff-failed");
	
	hipEventRecord(t0);
	//produce the indices
	StreamCompaction::Common::kernMapToBoolean<<<numblocks, blockSize>>> ( n, bool_buff, idata_buff);

	scan  (n, indices_buff, bool_buff);

	StreamCompaction::Common::kernScatter<<<numblocks, blockSize>>>( n, odata_buff, idata_buff,  bool_buff,  indices_buff);
	
	hipEventRecord(t2);
	hipEventSynchronize(t2);	
	hipEventElapsedTime(&tmpt, t0, t2);
	milliscs += tmpt;

	//GPU-->CPU
	hipMemcpy(odata,odata_buff,n*sizeof(int),hipMemcpyDeviceToHost);

	//for (int i =0; i< n; i++){
	//	n_remaing+=bool_buff[i];
	//}
	hipMemcpy(&n_remaing,indices_buff+n-1,sizeof(int),hipMemcpyDeviceToHost);
	int extra;
	hipMemcpy(&extra, bool_buff + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(idata_buff);
	hipFree(odata_buff);
	hipFree(bool_buff);
	hipFree(indices_buff);
	return n_remaing + extra;
}
int getMyCompactIndices(int n, int *dev_indices, int * dev_bools, const int *dev_data){//using dev pointer, not CPU !!
	int n_remaing = 0;  
	dim3 numblocks(std::ceil((double)n / blockSize));
/*	hipMemset(dev_bools, 0, n*sizeof(int));
	hipMemset(dev_indices, 0, n*sizeof(int));
	checkCUDAError("hipMemset-dev_bools-dev_indices failed"); 	*/ 
	//produce the indices
	StreamCompaction::Common::kernMapToBoolean << <numblocks, blockSize >> > (n, dev_bools, dev_data);

	scanOnDevice(n, dev_indices, dev_bools);
	 
	hipMemcpy(&n_remaing, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	int extra;
	hipMemcpy(&extra, dev_bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
 
	return n_remaing + extra;
	 
}

__global__ void kernMapPathsToBoolean(int n, int *bools, const PathSegment *paths) {
	// TODO
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n){
		bools[index] = paths[index].remainingBounces != 0;
	}
}
__global__ void kernPathsScatter(int n, PathSegment *odata,
	const PathSegment *idata, const int *bools, const int *indices) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index >= n) return;
	if (bools[index]){
		odata[indices[index]] = idata[index];
	}
}


int compactPaths(int n, PathSegment * odata_buff, int * bool_buff, int * indices_buff, PathSegment *paths){
 


	int n_remaing = 0;
 
	dim3 numblocks(std::ceil((double)n / blockSize));
	//
 
 

	 
	hipMemcpy(odata_buff, paths, n* sizeof(PathSegment), hipMemcpyDeviceToDevice);
	checkCUDAError("hipMemcpy-odata_buff-failed");

	 ;
	//produce the indices
	kernMapPathsToBoolean << <numblocks, blockSize >> > (n, bool_buff, paths);

	scanOnDevice(n, indices_buff, bool_buff);

	kernPathsScatter << <numblocks, blockSize >> >(n, odata_buff, paths, bool_buff, indices_buff);

 

	//GPU-->GPU
	hipMemcpy(paths, odata_buff, n*sizeof(PathSegment), hipMemcpyDeviceToDevice);

 
	hipMemcpy(&n_remaing, indices_buff + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	int extra;
	hipMemcpy(&extra, bool_buff + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	 
 
	return n_remaing + extra;
}
}
}
